#include "hip/hip_runtime.h"
#include "defines.hpp"

__global__ void goodPixels(int *d_data, int *d_pCandidateOffsets, int *d_aux, int *d_nCandidates, int width,int d_max){

	int row= blockIdx.y*dimOfBlock+threadIdx.y;
	int col= blockIdx.x*dimOfBlock+threadIdx.x;
	int id=row*width+col;

	if(d_data[id]>=d_max){
		int temp = atomicAdd(&d_nCandidates[0],1);
		//d_aux[temp]=d_data[id];
		d_pCandidateOffsets[temp]=id;

		}
}
