#include "hip/hip_runtime.h"
#include "defines.hpp"

extern __shared__ unsigned int Hs[];
__global__ void GradientCalculation(int width,int height, const unsigned char *input,
		unsigned int *pixHist_, const int stop, int *d_p, int *d_q, int *d_pq,
		int BINS, int BINSp, int R){//se le debe pasar el offset*

	int row= blockIdx.y*dimOfBlock+threadIdx.y;
	int col= blockIdx.x*dimOfBlock+threadIdx.x;
	int id=row*width+col;
	int temp=0;
	int size=width*height;

	///////////////////
	// Block and thread index
    const int bx = blockIdx.x;
    //const int tx = threadIdx.x;

    const int tx = threadIdx.y*dimOfBlock+threadIdx.x;

	//ty*blockdim+tx

    // Offset to per-block sub-histograms
    const unsigned int off_rep = BINSp * (tx % R);//total 11 bins, el modulo se multiplica por el tamaño del bin

	///////////////////
	for(int pos = tx; pos < BINSp*R; pos += blockDim.x*blockDim.y) Hs[pos] = 0;
	__syncthreads();        // Intra-block synchronization

	__shared__ unsigned char s_input[dimOfBlock][dimOfBlock];
	s_input[threadIdx.y][threadIdx.x]=input[id];

	__syncthreads();
	d_p[id]=0;
	d_q[id]=0;
	d_pq[id]=0;


	int p,q;
	if(id>=width+1 && id<stop  ){


		if(threadIdx.x==0){
			p = int(s_input[threadIdx.y][threadIdx.x+1]) - int(input[id - 1]);
		}
		else if(threadIdx.x==dimOfBlock-1){
			p = int(input[id + 1]) - int(s_input[threadIdx.y][threadIdx.x-1]);
		}
		else
			p = int(s_input[threadIdx.y][threadIdx.x+1]) - int(s_input[threadIdx.y][threadIdx.x-1]);

		//Columnas
		if(threadIdx.y==0){
			q = int(s_input[threadIdx.y+1][threadIdx.x]) - int(input[id - width]);
		}
		else if(threadIdx.y==dimOfBlock-1){
			q = int(input[id + width]) - int(s_input[threadIdx.y-1][threadIdx.x]);
		}
		else
			q = int(s_input[threadIdx.y+1][threadIdx.x]) - int(s_input[threadIdx.y-1][threadIdx.x]);

		d_p[id] = p * p;
		d_q[id] = q * q;
		d_pq[id] = p * q;

		/*temp = (abs(d_pq[id]));

		if(temp<=1023)
                	atomicAdd(&Hs[off_rep + temp], 1);
                else
                	atomicAdd(&Hs[off_rep + 1023], 1);*/
		}
	 /*__syncthreads();      // Intra-block synchronization

        // Merge per-block histograms and write to global memory

	for(int pos = tx; pos < BINS; pos += blockDim.x*blockDim.y){
		unsigned int sum = 0;
		for(int base = 0; base < BINSp*R; base += BINSp)
		        sum += Hs[base + pos];
		// Atomic addition in global memory
		atomicAdd(pixHist_ + pos, sum);
		}*/

}
